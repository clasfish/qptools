#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include "cumatrix_base.h"
#include "cumatrix_util.h"
#include "cuqp.h"
namespace py = pybind11;


py::buffer_info cumatrix_toBuffer(const cumatrix* A){
    double* a = new double[A->size];
    hipMemcpy(a, A->begin, A->size * sizeof(double), hipMemcpyDeviceToHost);
    return py::buffer_info(
        a,
        sizeof(double),
        py::format_descriptor<double>::format(),
        2,
        {A->nrows, A->ncols},
        {sizeof(double), sizeof(double)*A->nrows}
    );
}

cumatrix* cumatrix_fromBuffer(py::buffer buf){
    py::buffer_info info = buf.request();
    if(info.format != py::format_descriptor<double>::format())
        throw std::runtime_error("Incompatible format: expected a double array");
    if(info.ndim == 1){
        const int size=info.shape[0];
        std::cout << size << std::endl;
        cumatrix *A = new cumatrix(size, 1);
        double *a = new double[size], *iter=a, *end=a+size;
        double *iter0 = static_cast<double *>(info.ptr);
        for(; iter<end; iter++, iter0++) *iter = *iter0;
        hipMemcpy(A->begin, a, size*sizeof(double), hipMemcpyHostToDevice);
        return A;
    }else if(info.ndim == 2){
        int i, j;
        const int nrows=info.shape[0], ncols=info.shape[1], size=nrows*ncols, stride0=info.strides[0]/sizeof(double), stride1=info.strides[1]/sizeof(double);
        cumatrix *A = new cumatrix(nrows, ncols);
        double *a = new double[size], *itera=a;
        const double *b = static_cast<double *>(info.ptr), *iterb;
        for(j=0;j<ncols;j++){
            iterb = b + j * stride1;
            for(i=0;i<nrows;i++,itera++,iterb+=stride0) *itera = *iterb;
        }
        hipMemcpy(A->begin, a, size*sizeof(double), hipMemcpyHostToDevice);
        return A;
    }else throw std::runtime_error("Incompatible buffer dimension: " + std::to_string(info.ndim));
}

std::string cumatrix_repr(const cumatrix* a){
    return "<qptools.cumatrix of size (" +
        std::to_string(a->nrows) + "," + std::to_string(a->ncols) + 
        ")>";
}


PYBIND11_MODULE(cudacore, m){
    m.def("matrix_fromBuffer", &cumatrix_fromBuffer);
    py::class_<cumatrix>(m, "cumatrix", py::buffer_protocol())
        .def(py::init<int, int>())
        .def(py::init<int, int, double>())
        .def_readonly("nrows", &cumatrix::nrows)
        .def_readonly("ncols", &cumatrix::ncols)
        .def_readonly("size", &cumatrix::size)
        .def("display", &cumatrix::display)
        .def("_display", &cumatrix::_display, py::arg("len"))
        .def("__repr__", &cumatrix_repr)
        .def("fill", &cumatrix::fill)
        .def("sum", &cumatrix::sum)
        .def("min", &cumatrix::min)
        .def_buffer(&cumatrix_toBuffer);
    py::class_<CublasHandle>(m, "CublasHandle")
        .def(py::init<>());
    py::class_<CusolverHandle>(m, "CusolverHandle")
        .def(py::init<>());
    py::class_<cuqp1>(m, "cuqp1")
        .def(
            py::init<const CublasHandle&, const CusolverHandle&, const cumatrix*, const cumatrix*, const cumatrix*, const cumatrix*, const cumatrix*, const cumatrix*>(),
            py::arg("_cublas_handle"),
            py::arg("_cusolver_handle"),
            py::arg("P"),
            py::arg("q") = nullptr,
            py::arg("lb") = nullptr,
            py::arg("rb") = nullptr,
            py::arg("G") = nullptr,
            py::arg("h") = nullptr
        )
        .def("solve", &cuqp1::solve);
}