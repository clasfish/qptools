#include "matrix_util.cuh"


CublasHandle::CublasHandle(){
    hipblasCreate(&handle);
}

CublasHandle::~CublasHandle(){
    hipblasDestroy(handle);
}

CusolverHandle::CusolverHandle(){
    hipsolverDnCreate(&handle);
}

CusolverHandle::~CusolverHandle(){
    hipsolverDnDestroy(handle);
}
