#include <iostream>
#include "cumatrix_util.h"


CublasHandle::CublasHandle(){
    hipblasCreate(&handle);
}

CublasHandle::~CublasHandle(){
    hipblasDestroy(handle);
}

CusolverHandle::CusolverHandle(){
    hipsolverDnCreate(&handle);
}

CusolverHandle::~CusolverHandle(){
    hipsolverDnDestroy(handle);
}
