#include <iostream>
#include "cumatrix_util.h"


void check_status(hipblasStatus_t status){
    if (status != HIPBLAS_STATUS_SUCCESS){
        std::cerr << "cublas initialized failed:" << status << std::endl;
        exit(1);
    }
}

hipblasHandle_t create_handle(){
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    check_status(status);
    return handle;
}

